#include "hip/hip_runtime.h"
/*
*       Parallels and Distributed Systems Exercise 3
*       v0. Sequential version of Ising Model
*       Author:Michael Karatzas
*       AEM:9137
*/
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "ising.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
//The max threads per block for my gpu (gt 540m) is 1024 = 32*32 (1024 are run by a single processor)
#define BLOCK_DIM_X 32
#define BLOCK_DIM_Y 32
//In my gpu there 2(MPs)*48(SPs)=96 sqrt(96)>9 => grid dimensions:
#define GRID_DIM_X 9
#define GRID_DIM_Y 9

//Functions Declaration
__global__
void nextStateCalculation(int *Gptr,int *newMat, double * w , int n);

__device__ __forceinline__
void getTheSpin(int * Lat,int * newLat, double * weights , int n, int rowIndex,int colIndex);

// __host__
// void checkKernelConfiguration

void pointer_swap(int **a , int **b);


///Functions Definition
void ising( int *G, double *w, int k, int n){

  int * d_G, *secondG, *d_secondG;
  double * d_w;

  //Allocate and Get the G Matrix in the Device
  hipMalloc((void **)&d_G, (size_t)sizeof(int)*n*n);
  hipMemcpy(d_G, G, (size_t)sizeof(int)*n*n, hipMemcpyHostToDevice);

  //Allocate and Get the Weights Matrix in the Device
  hipMalloc((void **)&d_w, (size_t)sizeof(double)*5*5);
  hipMemcpy(d_w, w, (size_t)sizeof(double)*5*5, hipMemcpyHostToDevice);

  //The second Matrix We use,allocation in CPU(host) and GPU(device)
  secondG= (int *)malloc((size_t)sizeof(int)*n*n);
  hipMalloc((void **)&d_secondG, (size_t)sizeof(int)*n*n);

  //check for valid Kernel Configuration

  //Evolving the model for k steps
  for(int i=0 ; i<k ;i++){


    //grid in order one thread to compute a block of moments.
    //dim3 BlockDim(BLOCK_SIZE_1D,BLOCK_SIZE_1D);

    dim3 dimBlock(BLOCK_DIM_X,BLOCK_DIM_Y);
    dim3 dimGrid(GRID_DIM_X,GRID_DIM_Y);
    //Check for valid kernel configuration

    nextStateCalculation<<<dimGrid,dimBlock>>>(d_G,d_secondG,d_w,n);
    hipMemcpy(secondG,d_secondG,(size_t)sizeof(int)*n*n,hipMemcpyDeviceToHost);
    // //checkErrors
    // printf("%s\n", hipGetErrorString(hipGetLastError()));

    //Swapping the pointers between the two Matrices.
    pointer_swap(&G,&secondG);
    //Update data in device after the pointer swap.
    hipMemcpy(d_G, G, (size_t)sizeof(int)*n*n, hipMemcpyHostToDevice);
    hipMemcpy(d_secondG, secondG, (size_t)sizeof(int)*n*n, hipMemcpyHostToDevice);




  }

  //Getting the right values to the initial Lattice matrix for odd number of spots
  if((k%2)!=0){
    memcpy (secondG, G, (size_t)sizeof(int)*n*n);
    //Freeing memory space I dont need from CPU and GPU to avoid memory leaks.
    free(G);
    hipFree(d_G);
    hipFree(d_secondG);
  }
  else{
    //Freeing memory space I dont need from CPU and GPU to avoid memory leaks.
    free(secondG);
    hipFree(d_G);
    hipFree(d_secondG);
  }
}
__global__
void nextStateCalculation(int *Gptr,int *newMat, double * w , int n){
      int strideX = blockDim.x *gridDim.x ;
      int strideY = blockDim.y *gridDim.y ;
      int index_X = threadIdx.x +blockDim.x*blockIdx.x;
      int index_Y = threadIdx.y +blockDim.y*blockIdx.y;


      for(int i=index_X;i<n ;i+=strideX){
        for(int j=index_Y; j<n;j+=strideY){
          getTheSpin(Gptr,newMat,w,n,i,j);
        }
      }
}

void getTheSpin(int * Lat,int * newLat, double * weights , int n, int rowIndex,int colIndex){
  // int rowIndex= index/n;
  // int colIndex= index%n;

  double total=0;
  int idxR,idxC;
  //Getting the total for a certain spin.
  for(int i=rowIndex-2;i<rowIndex+3;i++ ){
    for(int j=colIndex-2;j<colIndex+3;j++ ){
      if((i==rowIndex) && (j==colIndex))
        continue;

      //using modulus arithmetic for handle the edges
      //Getting the modulus from the remainder in negative values of Cmodulus operator
      idxR= (i % n + n) % n ;
      idxC= (j % n + n) % n ;

      total+=Lat[ idxR*n + idxC] *weights[(2+i-rowIndex)*5 + (2+j-colIndex)];
    }
  }

  //Checking the conditions
  //if (total ==0), with taking into account possible floating point errors
  if( (total<1e-6)  &&  (total>(-1e-6)) ){
    newLat[rowIndex*n+colIndex]=Lat[rowIndex*n+colIndex];
  }
  else if(total<0){
    newLat[rowIndex*n+colIndex]=-1;
  }
  else if(total>0){
    newLat[rowIndex*n+colIndex]=1;
  }

}

void pointer_swap(int **a , int **b){
  int * temp=*a;
  *a=*b;
  *b=temp;
}
