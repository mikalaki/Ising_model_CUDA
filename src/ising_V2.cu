#include "hip/hip_runtime.h"
/*
*       Parallels and Distributed Systems Exercise 3
*       v2. CUDA modified ising model ,grid and block computes the magnetic moments.
*       Author:Michael Karatzas
*       AEM:9137
*/
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "ising.h"
#include "essentials.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
//The max threads per block for my gpu (gt 540m) is 1024 = 32*32 (1024 are run by a single processor)
//(Preferably:set BLOCK_DIM_X and BLOCK_DIM_Y a multiple of 4)
#define BLOCK_DIM_X 32
#define BLOCK_DIM_Y 32
#define GRID_DIM_X 9
#define GRID_DIM_Y 9

//Functions Declaration
__global__
void nextStateCalculation(int *Gptr,int *newMat, double * w , int n);

__device__ __forceinline__
void getTheSpin(int * Lat,int * newLat, double * weights , int n, int rowIndex,int colIndex);



///Functions Definition
void ising( int *G, double *w, int k, int n){

  int * d_G, *d_secondG;
  double * d_w;

  //Allocate and Get the G Matrix in the Device
  if(   hipMalloc((void **)&d_G, (size_t)sizeof(int)*n*n)     != hipSuccess){
    printf("Couldn't allocate memory in device (GPU) !");
    exit(1);
  }
  hipMemcpy(d_G, G, (size_t)sizeof(int)*n*n, hipMemcpyHostToDevice);

  //Allocate and Get the Weights Matrix in the Device
  if(  hipMalloc((void **)&d_w, (size_t)sizeof(double)*5*5)   != hipSuccess){
    printf("Couldn't allocate memory in device (GPU) !");
    exit(1);
  }
  hipMemcpy(d_w, w, (size_t)sizeof(double)*5*5, hipMemcpyHostToDevice);

  //The second Matrix We use,allocation only in GPU(device)
  if(hipMalloc((void **)&d_secondG, (size_t)sizeof(int)*n*n) != hipSuccess){
    printf("Couldn't allocate memory in device (GPU) !");
    exit(1);
  }


  //Evolving the model for k steps
  for(int i=0 ; i<k ;i++){


    //grid in order one thread to compute a block of moments.
    //dim3 BlockDim(BLOCK_SIZE_1D,BLOCK_SIZE_1D);

    dim3 dimBlock(BLOCK_DIM_X,BLOCK_DIM_Y);
    dim3 dimGrid(GRID_DIM_X,GRID_DIM_Y);


    nextStateCalculation<<<dimGrid,dimBlock>>>(d_G,d_secondG,d_w,n);
    hipDeviceSynchronize();

    //Swapping the pointers between the two Matrices in device
    pointer_swap(&d_G,&d_secondG);

    //Passing updated values of G matrix in the CPU.
    hipMemcpy(G,d_G,(size_t)sizeof(int)*n*n,hipMemcpyDeviceToHost);


  }

  //Freeing memory space I dont need from GPU to avoid memory leaks.
  hipFree(d_G);
  hipFree(d_secondG);
  hipFree(d_w);

}
__global__
void nextStateCalculation(int *Gptr,int *newMat, double * w , int n){
      int strideX = blockDim.x *gridDim.x ;
      int strideY = blockDim.y *gridDim.y ;
      int index_X = threadIdx.x +blockDim.x*blockIdx.x;
      int index_Y = threadIdx.y +blockDim.y*blockIdx.y;


      for(int i=index_Y;i<n ;i+=strideY){
        for(int j=index_X; j<n;j+=strideX){
          getTheSpin(Gptr,newMat,w,n,i,j);
        }
      }
}
__device__ __forceinline__
void getTheSpin(int * Lat,int * newLat, double * weights , int n, int rowIndex,int colIndex){


  double total=0;
  int idxR,idxC;
  //Getting the total for a certain spin.
  for(int i=rowIndex-2;i<rowIndex+3;i++ ){
    for(int j=colIndex-2;j<colIndex+3;j++ ){
      if((i==rowIndex) && (j==colIndex))
        continue;

      //using modulus arithmetic for handle the edges
      //Getting the modulus from the remainder in negative values of Cmodulus operator
      idxR= (i + n) % n ;
      idxC= (j + n) % n ;

      total+=Lat[ idxR*n + idxC] *weights[(2+i-rowIndex)*5 + (2+j-colIndex)];
    }
  }

  //Checking the conditions
  //if (total ==0), with taking into account possible floating point errors
  if( (total<1e-6)  &&  (total>(-1e-6)) ){
    newLat[rowIndex*n+colIndex]=Lat[rowIndex*n+colIndex];
  }
  else if(total<0){
    newLat[rowIndex*n+colIndex]=-1;
  }
  else if(total>0){
    newLat[rowIndex*n+colIndex]=1;
  }

}
