#include "hip/hip_runtime.h"
/*
*       Parallels and Distributed Systems Exercise 3
*       v2. CUDA modified ising model evolution ,grid and block computes the magnetic moments.
*       Author:Michael Karatzas
*       AEM:9137
*/
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "ising.h"
#include "essentials.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
//The max threads per block for my gpu (gt 540m) is 1024 so it must be BLOCK_DIM_X* BLOCK_DIM_Y<=1024
//(Preferably:set BLOCK_DIM_X and BLOCK_DIM_Y a multiple of 4)
#define BLOCK_DIM_X 24
#define BLOCK_DIM_Y 24
#define GRID_DIM_X 4
#define GRID_DIM_Y 4

//Functions'-kernels' Declarations
__global__
void nextStateCalculation(int *Gptr,int *newMat, double * w , int n, int * flag);

__device__ __forceinline__
void getTheSpin(int * Lat,int * newLat, double * weights , int n, int rowIndex,
  int colIndex, int * flag);



///Functions'-kernels' Definitions
void ising( int *G, double *w, int k, int n){

  //Flag for indicate if there was no changes in the lattice during a step,in order to terminate the evolving.
  int no_changes_flag;

  int * d_G, *d_secondG, *d_no_changes_flag;
  double * d_w;

  //Allocate memory for the no change flag in the Device
  if(   hipMalloc(&d_no_changes_flag, (size_t)sizeof(int))    != hipSuccess){
    printf("Couldn't allocate memory in device (GPU) !");
    exit(1);
  }

  //Allocate memory and "transfer" the G Matrix in the Device
  if(   hipMalloc((void **)&d_G, (size_t)sizeof(int)*n*n)     != hipSuccess){
    printf("Couldn't allocate memory in device (GPU) !");
    exit(1);
  }
  hipMemcpy(d_G, G, (size_t)sizeof(int)*n*n, hipMemcpyHostToDevice);

  //Allocate memory and "transfer" the Weights' Matrix in the Device
  if(  hipMalloc((void **)&d_w, (size_t)sizeof(double)*5*5)   != hipSuccess){
    printf("Couldn't allocate memory in device (GPU) !");
    exit(1);
  }
  hipMemcpy(d_w, w, (size_t)sizeof(double)*5*5, hipMemcpyHostToDevice);

  //Allocate memory for the second G matrix only in GPU(device)
  if(hipMalloc((void **)&d_secondG, (size_t)sizeof(int)*n*n) != hipSuccess){
    printf("Couldn't allocate memory in device (GPU) !");
    exit(1);
  }

  //grid and block dimensions in order one thread to compute a set of moments.
  dim3 dimBlock(BLOCK_DIM_X,BLOCK_DIM_Y);
  dim3 dimGrid(GRID_DIM_X,GRID_DIM_Y);

  //Evolving the model for k steps
  for(int i=0 ; i<k ;i++){

    /*no_changes_flag=1, indicates no change in the lattice, if there are changes
    nextStateCalculation() kernel will update its value.*/
    no_changes_flag=1;
    hipMemcpy(d_no_changes_flag, &no_changes_flag, (size_t)sizeof(int), hipMemcpyHostToDevice);

    //calling the nextStateCalculation() kernel
    nextStateCalculation<<<dimGrid,dimBlock>>>(d_G,d_secondG,d_w,n,d_no_changes_flag);
    hipDeviceSynchronize();

    //Swapping the pointers between the two Matrices in device
    pointer_swap(&d_G,&d_secondG);

    //The host get the value of the no changes flag as indication if no changes happened during the step.
    hipMemcpy(&no_changes_flag, d_no_changes_flag,  (size_t)sizeof(int), hipMemcpyDeviceToHost);
    //If there are no changes in the lattice we stop evolving the model
    if(no_changes_flag){
      break;
    }

  }

  //Passing updated values of G matrix in the host(CPU).
  hipMemcpy(G,d_G,(size_t)sizeof(int)*n*n,hipMemcpyDeviceToHost);


  //Freeing memory space I dont need from GPU to avoid memory leaks.
  hipFree(d_G);
  hipFree(d_secondG);
  hipFree(d_w);

}
__global__
void nextStateCalculation(int *Gptr,int *newMat, double * w , int n, int * flag){
      //The step of each thread
      int strideX = blockDim.x *gridDim.x ;
      int strideY = blockDim.y *gridDim.y ;

      //The unigue global indixes of the threads in the grid
      int index_X = threadIdx.x +blockDim.x*blockIdx.x;
      int index_Y = threadIdx.y +blockDim.y*blockIdx.y;

      //Each thread loops in order to compute the spin of its own points
      for(int i=index_Y;i<n ;i+=strideY){
        for(int j=index_X; j<n;j+=strideX){
          getTheSpin(Gptr,newMat,w,n,i,j,flag);
        }
      }
}
__device__ __forceinline__
void getTheSpin(int * Lat,int * newLat, double * weights , int n, int rowIndex,
  int colIndex, int * flag){


  double total=0;
  int idxR,idxC;

  //Calculating the Total influence for a certain spot
  for(int i=rowIndex-2;i<rowIndex+3;i++ ){
    for(int j=colIndex-2;j<colIndex+3;j++ ){
      if((i==rowIndex) && (j==colIndex))
        continue;

      //using modulus arithmetic for handle the boundaries' conditions
      //Getting the positive modulus
      idxR= (i + n) % n ;
      idxC= (j + n) % n ;

      //Total influence update
      total+=Lat[ idxR*n + idxC] *weights[(2+i-rowIndex)*5 + (2+j-colIndex)];
    }
  }

  //Checking the conditions in order to get the next state spin
  //if (total ==0), with taking into account possible floating point errors
  if( (total<1e-6)  &&  (total>(-1e-6)) ){
    newLat[rowIndex*n+colIndex]=Lat[rowIndex*n+colIndex];
  }
  //if change in a certain spot happens we update no change flag's value into 0.
  else if(total<0){
    //Checking if there is change in this certain spot
    if(Lat[rowIndex*n+colIndex]!=1)
      *flag=0;
    newLat[rowIndex*n+colIndex]=-1;
  }
  else if(total>0){
    //Checking if there is change in this certain spot
    if(Lat[rowIndex*n+colIndex]!=-1)
      *flag=0;
    newLat[rowIndex*n+colIndex]=1;
  }

}
