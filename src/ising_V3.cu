#include "hip/hip_runtime.h"
/*
*       Parallels and Distributed Systems Exercise 3
*       v3. CUDA modified ising model evolution,each block use block threads' shared memory.
*       Author:Michael Karatzas
*       AEM:9137
*/
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "ising.h"
#include "essentials.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
//The max threads per block for my gpu (gt 540m) is 1024 so it must be BLOCK_DIM_X* BLOCK_DIM_Y<=1024
//(Preferably:set BLOCK_DIM_X and BLOCK_DIM_Y a multiple of 4)
#define BLOCK_DIM_X 24
#define BLOCK_DIM_Y 24
#define GRID_DIM_X 4
#define GRID_DIM_Y 4
#define RADIUS 2

//Functions'-kernels' Declarations
__global__
void nextStateCalculation(int *Gptr,int *newMat, double * w , int n, int * flag);

__device__ __forceinline__
void getTheSpin(int * Lat,int * newLat, double * weights , int n, int lRowIndex,
  int lColIndex,int gRowIndex,int gColIndex, int * flag);



///Functions'-kernels' Definitions
void ising( int *G, double *w, int k, int n){

  //Flag for indicate if there was no changes in the lattice during a step,in order to terminate the evolving.
  int no_changes_flag;

  int * d_G,*d_secondG, *d_no_changes_flag;
  double * d_w;

  //Allocate memory for the no changes flag in the Device
  if(   hipMalloc(&d_no_changes_flag, (size_t)sizeof(int))    != hipSuccess){
    printf("Couldn't allocate memory in device (GPU) !");
    exit(1);
  }

  //Allocate memory and "transfer" the G Matrix in the Device
  if(   hipMalloc((void **)&d_G, (size_t)sizeof(int)*n*n)     != hipSuccess){
    printf("Couldn't allocate memory in device (GPU) !");
    exit(1);
  }
  hipMemcpy(d_G, G, (size_t)sizeof(int)*n*n, hipMemcpyHostToDevice);

  //Allocate memory and "transfer" the Weights Matrix in the Device
  if(  hipMalloc((void **)&d_w, (size_t)sizeof(double)*5*5)   != hipSuccess){
    printf("Couldn't allocate memory in device (GPU) !");
    exit(1);
  }
  hipMemcpy(d_w, w, (size_t)sizeof(double)*5*5, hipMemcpyHostToDevice);

  //Allocate memory for the second G matrix only in GPU(device)
  if(hipMalloc((void **)&d_secondG, (size_t)sizeof(int)*n*n) != hipSuccess){
    printf("Couldn't allocate memory in device (GPU) !");
    exit(1);
  }

  //Setting block's and grid's dimensions
  dim3 dimBlock(BLOCK_DIM_X,BLOCK_DIM_Y);
  dim3 dimGrid(GRID_DIM_X,GRID_DIM_Y);


  //Evolving the model for k steps
  for(int i=0 ; i<k ;i++){

    /*no_changes_flag=1, indicates no change in the lattice, if there are changes
    nextStateCalculation() kernel will update its value.*/
    no_changes_flag=1;
    hipMemcpy(d_no_changes_flag, &no_changes_flag, (size_t)sizeof(int), hipMemcpyHostToDevice);


    //calling the nextStateCalculation() kernel
    nextStateCalculation<<<dimGrid,dimBlock>>>(d_G,d_secondG,d_w,n,d_no_changes_flag);
    hipDeviceSynchronize();

    //Swapping the pointers between the two Matrices in device
    pointer_swap(&d_G,&d_secondG);

    //The host get the value of the no changes flag as indication if no changes happened during the step.
    hipMemcpy(&no_changes_flag, d_no_changes_flag,  (size_t)sizeof(int), hipMemcpyDeviceToHost);
    //If there are no changes in the lattice we stop evolving the model
    if(no_changes_flag){
      break;
    }

  }

  //Passing updated values of G matrix in the host(CPU).
  hipMemcpy(G,d_G,(size_t)sizeof(int)*n*n,hipMemcpyDeviceToHost);

  //Freeing memory space I don't need from GPU to avoid memory leaks.
  hipFree(d_G);
  hipFree(d_secondG);
  hipFree(d_w);

}
__global__
void nextStateCalculation(int *Gptr,int *newMat, double * w , int n, int * flag){
      /* The part of the G matrix that is needed to be read in the block shared memory
      are the spots that their spin is going to get computed by the block
      and two "offset" spot around every edgy spot, that will be needed for the spin
      computation as they are neighbots of the edgy spots  */

      //The part of the G matrix that will pass in the shared memory.
      __shared__ int sharedGpart[(BLOCK_DIM_X+2*RADIUS)  *  (BLOCK_DIM_Y+2*RADIUS)];

      //The number of columns of the shared G part
      int sharedNcols=(BLOCK_DIM_X+2*RADIUS) ;

      //matrix to store the shared weight matrices
      __shared__ double w_shared[25];

      //The step of each thread
      int strideX = blockDim.x *gridDim.x ;
      int strideY = blockDim.y *gridDim.y ;

      //The unigue global indixes of the threads in the grid
      int gIndex_X = threadIdx.x +blockDim.x*blockIdx.x;//global x index
      int gIndex_Y = threadIdx.y +blockDim.y*blockIdx.y;//global y index

      //The local (in the block) Index
      int lIndex_X=threadIdx.x+RADIUS;//local(in the block) x index
      int lIndex_Y=threadIdx.y+RADIUS;//local(in the block) y index

      //Accessing the spins in the global lattice and pass them in the shared matrix.
      for(int i=gIndex_Y; i<n +RADIUS ;i+=strideY){
        for(int j=gIndex_X; j<n +RADIUS;j+=strideX){

          //Every thread read its own element in shared memory
          sharedGpart[lIndex_Y*sharedNcols+lIndex_X]=Gptr[( (i + n)%n )*n + ( (j + n)%n )];

          //Accessing and read read in shared memory the 2 left and 2 right "offset" elements on each row
          if((threadIdx.x)<RADIUS){
            int sharedGAccessorX= (lIndex_Y)*sharedNcols+(lIndex_X -RADIUS);
            int GAccessorX=( (i + n)%n )*n+ ( ( (j-RADIUS)  + n) % n);
            sharedGpart[sharedGAccessorX]=Gptr[GAccessorX];

            sharedGAccessorX=(lIndex_Y)*sharedNcols+(lIndex_X+BLOCK_DIM_X);
            GAccessorX=( (i + n)%n )*n+( ( (j+BLOCK_DIM_X)  + n) % n);
            sharedGpart[sharedGAccessorX]=Gptr[GAccessorX];

            //Accessing and read in shared memory "corner offset" elements(each corner has 4 elements)
            if((threadIdx.y)<RADIUS){
              //1st corner (4 points, up and left)
              int sharedDiagAccessorX= (lIndex_Y -RADIUS)*sharedNcols +(lIndex_X-RADIUS);
              int GDiagAccessorX=( ( (i-RADIUS)  + n) % n)*n+( ( (j-RADIUS)  + n) % n);
              sharedGpart[sharedDiagAccessorX]=Gptr[GDiagAccessorX];

              //2nd diagonial (4 points, down and left)
              sharedDiagAccessorX= (lIndex_Y+BLOCK_DIM_Y)*sharedNcols +(lIndex_X-RADIUS);
              GDiagAccessorX=( ( (i+BLOCK_DIM_Y)  + n) % n)*n+( ( (j-RADIUS)  + n) % n);
              sharedGpart[sharedDiagAccessorX]=Gptr[GDiagAccessorX];

              //3rd corner (4 points, down and right)
              sharedDiagAccessorX= (lIndex_Y+BLOCK_DIM_Y)*sharedNcols +(lIndex_X+BLOCK_DIM_X);
              GDiagAccessorX=( ( (i+BLOCK_DIM_Y)  + n) % n)*n+( ( (j+BLOCK_DIM_X)  + n) % n);
              sharedGpart[sharedDiagAccessorX]=Gptr[GDiagAccessorX];

              //4rd diagonial (4 points, up and right)
              sharedDiagAccessorX= (lIndex_Y -RADIUS)*sharedNcols+(lIndex_X+BLOCK_DIM_X);
              GDiagAccessorX=( ( (i-RADIUS)  + n) % n)*n+( ( (j+BLOCK_DIM_X)  + n) % n);
              sharedGpart[sharedDiagAccessorX]=Gptr[GDiagAccessorX];
            }
          }

          //Accessing and read read in shared memory the 2 top and 2 bottom "offset" elements on each row
          if((threadIdx.y)<RADIUS){
            int sharedGAccessorY= (lIndex_Y-RADIUS)*sharedNcols+lIndex_X;
            int GAccessorY=( ( (i-RADIUS)  + n) % n)*n+( (j + n)%n );
            sharedGpart[sharedGAccessorY]=Gptr[GAccessorY];

            sharedGAccessorY=(lIndex_Y+BLOCK_DIM_Y)*sharedNcols+lIndex_X;
            GAccessorY=( ( (i+BLOCK_DIM_Y)  + n) % n)*n+( (j + n)%n );
            sharedGpart[sharedGAccessorY]=Gptr[GAccessorY];
          }

          /*Ιf (BLOCK_DIM_Y>=5) && (BLOCK_DIM_X>=5),we use shared memory  also for the weights matrix,
          I didn't implement it for smaller dimensions, because the benefit is very small anyway and it will
          make our code more complex .Also we choose BLOCK_DIM_X = BLOCK_DIM_Y =24 and BLOCK_DIM_X<5 or
          BLOCK_DIM_Y<5 aren't used in practice, so WE get the small benefit by transfering the weights' matrix.  */
          if((BLOCK_DIM_Y>=5) && (BLOCK_DIM_X>=5)){
            if(threadIdx.x<5 &&threadIdx.y<5)
              w_shared[threadIdx.x*5+ threadIdx.y]=w[threadIdx.x*5+ threadIdx.y];
          }



          //Here we synchronize the block threads in order Shared G values are
          //updated for each thread and w values are updated
          __syncthreads();

          if((i<n)&&(j<n)){
            if((BLOCK_DIM_Y>=5) && (BLOCK_DIM_X>=5))
              getTheSpin(sharedGpart,newMat,  w_shared,n,lIndex_Y, lIndex_X,i,j,flag);
            else //if((BLOCK_DIM_Y<5) && (BLOCK_DIM_X35))
              getTheSpin(sharedGpart,newMat,  w,n,lIndex_Y, lIndex_X,i,j,flag);

          }

          __syncthreads();

        }
      }

}
__device__ __forceinline__
void getTheSpin(int * Lat,int * newLat, double * weights , int n, int lRowIndex,int lColIndex,
int gRowIndex,int gColIndex, int * flag ){

  double total=0;
  //Calculating the Total influence for a certain spot, by scanning the block shared part of G.
  for(int i=lRowIndex-2;i<lRowIndex+3;i++ ){
    for(int j=lColIndex-2;j<lColIndex+3;j++ ){
      if((i==lRowIndex) && (j==lColIndex))
        continue;

      //Total influence update
      total+=Lat[ i*(BLOCK_DIM_X+2*RADIUS) + j] *weights[(2+i-lRowIndex)*5 + (2+j-lColIndex)];

    }
  }

  //Checking the conditions in order to get the next state spin
  //  if (total ==0), with taking into account possible floating point errors
  if( (total<1e-6)  &&  (total>(-1e-6)) ){
    newLat[(gRowIndex)*n+(gColIndex)]=Lat[lRowIndex*(BLOCK_DIM_X+2*RADIUS)+lColIndex];
  }
  //if change in a certain spot happens we update no change flag's value into 0.
  else if(total<0){
    //Checking if there is change in this certain spot
    if(Lat[lRowIndex*(BLOCK_DIM_X+2*RADIUS)+lColIndex]!=1)
      *flag=0;
    newLat[(gRowIndex)*n+(gColIndex)]=-1;
  }
  else if(total>0){
    //Checking if there is change in this certain spot
    if(Lat[lRowIndex*(BLOCK_DIM_X+2*RADIUS)+lColIndex]!=1)
      *flag=0;
    newLat[(gRowIndex)*n+(gColIndex)]=1;
  }

}
