#include "hip/hip_runtime.h"
/*
*       Parallels and Distributed Systems Exercise 3
*       v1. CUDA modified ising model evolution ,one thread computes a magnetic moment.
*       Author:Michael Karatzas
*       AEM:9137
*/
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "essentials.h"
#include "ising.h"
#include "hip/hip_runtime.h"
/*Setting the dimensions of the block,grid dimensions are computed int the programm
in order to have one thread per moment */
#define BLOCK_DIM_X 16
#define BLOCK_DIM_Y 16

//Functions'-Kernels' Declaration
__global__
void nextStateCalculation(int *Gptr,int *newMat, double * w , int n, int * flag);
__device__ __forceinline__
void getTheSpin(int * Lat,int * newLat, double * weights , int n, int rowIndex,
  int colIndex, int * flag);


///Functions'-kernels' Definitions
void ising( int *G, double *w, int k, int n){

  //Flag for indicate if there was no changes in the lattice during a step,in order to terminate the evolving.
  int no_changes_flag;

  int * d_G, *d_secondG, *d_no_changes_flag;
  double * d_w;


  //Allocate memory for the no changes flag in the Device
  if(   hipMalloc(&d_no_changes_flag, (size_t)sizeof(int))    != hipSuccess){
    printf("Couldn't allocate memory in device (GPU) !");
    exit(1);
  }


  //Allocate memory and "transfer" the given G Matrix in the Device
  hipMalloc((void **)&d_G, (size_t)sizeof(int)*n*n);
  if(   hipMalloc((void **)&d_G, (size_t)sizeof(int)*n*n)     != hipSuccess){
    printf("Couldn't allocate memory in device (GPU) !");
    exit(1);
  }
  hipMemcpy(d_G, G, (size_t)sizeof(int)*n*n, hipMemcpyHostToDevice);

  //Allocate memory and "transfer" the Weights' Matrix in the Device
  if(  hipMalloc((void **)&d_w, (size_t)sizeof(double)*5*5)   != hipSuccess){
    printf("Couldn't allocate memory in device (GPU) !");
    exit(1);
  }
  hipMemcpy(d_w, w, (size_t)sizeof(double)*5*5, hipMemcpyHostToDevice);

  //Allocate memory for the second G matrix only in GPU(device)
  if(hipMalloc((void **)&d_secondG, (size_t)sizeof(int)*n*n) != hipSuccess){
    printf("Couldn't allocate memory in device (GPU) !");
    exit(1);
  }

  /*grid that matches the ising Model (gridDimX*BLOCK_DIM_X <=n,gridDimY*BLOCK_DIM_Y <=n )
  configuration for 1 thread per moment.*/

  //Calculating the grid dimensions, in order to match the ising model.
  int gridDimX= (n+BLOCK_DIM_X -1)/BLOCK_DIM_X;
  int gridDimY= (n+BLOCK_DIM_Y -1)/BLOCK_DIM_Y;

  //Setting grid and block dimensions
  dim3 dimGrid(gridDimX,gridDimY);
  dim3 dimBlock(BLOCK_DIM_X,BLOCK_DIM_Y);


  //Evolving the model for k steps
  for(int i=0 ; i<k ;i++){

    /*no_changes_flag=1, indicates no change in the lattice, if there are changes
    nextStateCalculation() kernel will update its value.*/
    no_changes_flag=1;
    hipMemcpy(d_no_changes_flag, &no_changes_flag, (size_t)sizeof(int), hipMemcpyHostToDevice);

    //calling the nextStateCalculation() kernel
    nextStateCalculation<<<dimGrid,dimBlock>>>(d_G,d_secondG,d_w,n,d_no_changes_flag);
    hipDeviceSynchronize();

    //Swapping the pointers between the two Matrices in device
    pointer_swap(&d_G,&d_secondG);

    //The host get the value of the no changes flag as indication if no changes happened during the step.
    hipMemcpy(&no_changes_flag, d_no_changes_flag,  (size_t)sizeof(int), hipMemcpyDeviceToHost);
    //If there are no changes in the lattice we stop evolving the model
    if(no_changes_flag){
      break;
    }


  }
  //Passing updated values of G matrix in the host(CPU).
  hipMemcpy(G,d_G,(size_t)sizeof(int)*n*n,hipMemcpyDeviceToHost);


  //Freeing memory space I dont need from GPU to avoid memory leaks.
  hipFree(d_G);
  hipFree(d_secondG);
  hipFree(d_w);

}

__global__
void nextStateCalculation(int *Gptr,int *newMat, double * w , int n, int * flag){
    //The unigue global indixes of the threads.
    int index_X = threadIdx.x +blockDim.x*blockIdx.x;
    int index_Y = threadIdx.y +blockDim.y*blockIdx.y;

    //getting rid of the odd trheads
    if((index_X < n) &&(index_Y < n) ){

      //Get each thread calcute spin of its spot
      getTheSpin(Gptr,newMat,w,n,index_X,index_Y, flag);
    }
}
__device__ __forceinline__
void getTheSpin(int * Lat,int * newLat, double * weights , int n, int rowIndex,int colIndex, int * flag){


  double total=0;
  int idxR,idxC;

  //Calculating the Total influence for a certain spot.
  for(int i=rowIndex-2;i<rowIndex+3;i++ ){
    for(int j=colIndex-2;j<colIndex+3;j++ ){
      if((i==rowIndex) && (j==colIndex))
        continue;

      //using modulus arithmetic for handle the boundaries' conditions
      //Getting the positive modulus
      idxR= (i + n) % n ;
      idxC= (j + n) % n ;

      //Total influence update
      total+=Lat[ idxR*n + idxC] *weights[(2+i-rowIndex)*5 + (2+j-colIndex)];
    }
  }

  //Checking the conditions in order to get the next state spin
  //if (total ==0), taking into account possible floating point errors
  if( (total<1e-6)  &&  (total>(-1e-6)) ){
    newLat[rowIndex*n+colIndex]=Lat[rowIndex*n+colIndex];
  }
  //if change in a certain spot happens we update no changes flag's value into 0.
  else if(total<0){
    //Checking if there is change in this certain spot
    if(Lat[rowIndex*n+colIndex]!=-1)
      *flag=0;
    newLat[rowIndex*n+colIndex]=-1;
  }
  else if(total>0){
    //Checking if there is change in this certain spot
    if(Lat[rowIndex*n+colIndex]!=1)
      *flag=0;
    newLat[rowIndex*n+colIndex]=1;
  }

}
